#include "hip/hip_runtime.h"
#include "graphics.h"

__global__ void create_Grid_points_only_kernel(float4 *dDptr, float *dEz_float, int width, int height, float delta_x, float delta_y) {
	// This kernel turns the blank vertex spatial array into a properly formatted 
	// array of vertices that can be drawn during the display call.  
	__shared__ float dev_delta_x;
	__shared__ float dev_delta_y;
	dev_delta_x = delta_x;							// x-resolution
	dev_delta_y = delta_y;
	
	int offset = blockIdx.x * blockDim.x + threadIdx.x;	// Calculate linear offset for 1-D unrolled array
	int j = offset / width;							// Creates a virtual row index for the 1-D case, needed for odd/even row check
	int i = offset % width;							// Keeping both offset and i for clarity
	float x, y;

	// calculate uv coordinates
	float u = i / (float)width;
	float v = j / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;
	float w = 0.0f;

	x = -1.0 + i * dev_delta_x;
	y = -1.0 + j * dev_delta_y;
	u = x;
	w = y;
	v = 0.0f;

	// write output vertex
	if (offset < (width*height)) dDptr[offset] = make_float4(u, w, v, 1.0f);
}

__global__ void find_min_and_max_on_gpu(int nblocks, float* field, 
										float* minimum_field_value, 
										float* maximum_field_value)
{
	__shared__ float minarr[1024];
	__shared__ float maxarr[1024];

	int i = threadIdx.x;
	int nTotalThreads = blockDim.x;

	minarr[i] = field[i];
	maxarr[i] = minarr[i];
	for (int j = 1; j<nblocks; j++)
	{
		minarr[i + nTotalThreads] = field[i + nTotalThreads*j];
		if (minarr[i] > minarr[i + nTotalThreads])
			minarr[i] = minarr[i + nTotalThreads];

		if (maxarr[i] < minarr[i + nTotalThreads])
			maxarr[i] = minarr[i + nTotalThreads];
		__syncthreads();
	}
	__syncthreads();

	while (nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		if (threadIdx.x < halfPoint)
		{
			float temp = minarr[i + halfPoint];

			if (temp < minarr[i]) minarr[i] = temp;

			temp = maxarr[i + halfPoint];
			if (temp > maxarr[i]) maxarr[i] = temp;
		}
		__syncthreads();
		nTotalThreads = (nTotalThreads >> 1);
	}
	if (i == 0)
	{
		minimum_field_value[0] = minarr[0];
		maximum_field_value[0] = maxarr[0];
	}
}

__global__ void create_image_on_gpu_vbo_kernel(uchar4 *colorPos, float4 *pos, float* Ez, int M, float minval, float maxval, int time, int width, int height)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x;	// Calculate linear offset for 1-D unrolled array
	int j = offset / width;							// Creates a virtual row index for the 1-D case, needed for odd/even row check
	int i = offset % width;							// Keeping both offset and i for clarity
	float F;
	F = Ez[offset] - minval;
	F = F / (maxval - minval);
	
	// calculate uv coordinates
	float u = i / (float)width;
	float v = j / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;
	
	colorPos[offset].w = 0;
	colorPos[offset].x = 255.f *0.7*(F);
	colorPos[offset].y = 255.f *0.3*(F);
	colorPos[offset].z = 255.f *0.5*(F);
		
	// The following block is not used.  It is here to demonstrate visualization in the z axis instead of a color.
	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;
	// write output vertex
	//pos[offset] = make_float4(u, w, v, 1.0f);	
}

void createImageOnGpuVbo()	// argument g_odata is the float Ez field 
{												// array, coming in as a device pointer
	// The following are for find_min_and_max_on_gpu
	dim3 gridm = dim3(1, 1, 1);
	dim3 blockm = dim3(TILE_SIZE*TILE_SIZE, 1, 1);
	int  nblocks = g->M / TILE_SIZE * g->N / TILE_SIZE;
		
	// The following are for create_image_on_gpu_kernel
	int TILE_SQUARED = TILE_SIZE * TILE_SIZE;
	int Bx = (TILE_SQUARED - 1 + g->nCells) / TILE_SQUARED;
	dim3 BLK(Bx, 1, 1);									// Grid-block dimension for the 1-D case
	dim3 THD(TILE_SQUARED, 1, 1);							// Thread-block dimension for the 1-D case

	float minval;
	float maxval;
	float *dvF;

	//if (show_Ez) dvF = dev_ez_float; else dvF = dev_hx_float;
	dvF = dev_ez_float;

	find_min_and_max_on_gpu << < gridm, blockm >> >(nblocks, dvF, dvminimum_field_value, dvmaximum_field_value);

	hipMemcpy(&minval, dvminimum_field_value, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&maxval, dvmaximum_field_value, sizeof(float), hipMemcpyDeviceToHost);

	if (minval > 0.0) minval = 0.0;
	if (maxval < 0.0) maxval = 0.0;
	if (abs(minval) > maxval) maxval = -minval; else minval = -maxval;
	if (minval < global_min_field) global_min_field = minval;
	if (maxval > global_max_field) global_max_field = maxval;

	//hipMemcpy(g->ez_float, dvF, g->nCells*sizeof(float), hipMemcpyDeviceToHost);
	//for (int i = 0; i < g->nCells; i++)
	//	if (g->ez_float[i] > 0.001f || g->ez_float[i] < -0.001f)
	//		printf("ez_float[%i] = %f\n", i, g->ez_float[i]);

	//minval = -1.0;	maxval = 1.0;	global_min_field = -1.0; global_max_field = 1.0;
	//the following kernel now takes a uchar4 array, not uint
	create_image_on_gpu_vbo_kernel << < BLK, THD >> >(cptr, dptr, dvF, g->M, global_min_field, global_max_field, g->time, g->M, g->N);
}
void create_Grid_points_only(float4* dDptr, float *dEz_float)
{
	// This function and kernel get called only once to create the spatial portion
	// of the vertex buffer object.  The colors will be updated seperately each loop.
	// Calculate CUDA grid dimensions.  Block dimension fixed at 32x32 threads
	int TILE_SQUARED = TILE_SIZE * TILE_SIZE;
	int Bx = (TILE_SQUARED - 1 + g->nCells) / TILE_SQUARED;
	dim3 BLK(Bx, 1, 1);									// Grid-block dimension for the 1-D case
	dim3 THD(TILE_SQUARED, 1, 1);							// Thread-block dimension for the 1-D case
	float delta_x = 2.f / ((float)g->M);	// x-resolution of grid in screen space (uv)
	float delta_y = 2.f / ((float)g->M);	// y-resolution of grid in screen space (uv)

	create_Grid_points_only_kernel << < BLK, THD >> >(dDptr, dEz_float, g->M, g->N, delta_x, delta_y);		// kernel call
}